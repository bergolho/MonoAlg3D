#include "hip/hip_runtime.h"
#include <stddef.h>
#include "../monodomain/constants.h"
#include "model_gpu_utils.h"

#include "ten_tusscher_3_Fig4b.h"

#define ENDO


extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) {

    print_to_stdout_and_file("Using ten Tusscher 3 GPU model\n");

    // execution configuration
    const int GRID  = (num_volumes + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t size = num_volumes*sizeof(real);

    check_cuda_error(hipMallocPitch((void **) &(*sv), &pitch_h, size, (size_t )NEQ));
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(pitch), &pitch_h, sizeof(size_t)));

    kernel_set_model_inital_conditions <<<GRID, BLOCK_SIZE>>>(*sv, num_volumes);

    check_cuda_error( hipPeekAtLastError() );
    hipDeviceSynchronize();
    return pitch_h;

}

extern "C" SOLVE_MODEL_ODES_GPU(solve_model_odes_gpu) {

    // execution configuration
    const int GRID  = ((int)num_cells_to_solve + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t stim_currents_size = sizeof(real)*num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t)*num_cells_to_solve;

    real *stims_currents_device;
    check_cuda_error(hipMalloc((void **) &stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));

    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **) &cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }

    // Default values for a healthy cell ///////////
    real atpi = 6.8f;
    real Ko = 5.4f;
    real Ki_mult = 1.0f;
    real acidosis = 0.0;
    real K1_mult = 1.0f;
    ////////////////////////////////////

    real *fibrosis_device;
    real *fibs = NULL;

    if(extra_data) {
        atpi = ((real*)extra_data)[0]; //value
        Ko = ((real*)extra_data)[1]; //value
        Ki_mult = ((real*)extra_data)[2]; //value
        K1_mult = ((real*)extra_data)[3]; //value
        acidosis = ((real*)extra_data)[4]; //value
        fibs = ((real*)extra_data) + 5; //pointer

        extra_data_bytes_size = extra_data_bytes_size-5*sizeof(real);
    }
    else {
        extra_data_bytes_size = num_cells_to_solve*sizeof(real);
        fibs = (real*)calloc(num_cells_to_solve, sizeof(real));
    }

    check_cuda_error(hipMalloc((void **) &fibrosis_device, extra_data_bytes_size));
    check_cuda_error(hipMemcpy(fibrosis_device, fibs, extra_data_bytes_size, hipMemcpyHostToDevice));

    solve_gpu<<<GRID, BLOCK_SIZE>>>(dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve, num_steps, fibrosis_device, atpi, Ko, Ki_mult, K1_mult, acidosis);

    check_cuda_error( hipPeekAtLastError() );

    check_cuda_error(hipFree(stims_currents_device));
    check_cuda_error(hipFree(fibrosis_device));

    if(cells_to_solve_device) check_cuda_error(hipFree(cells_to_solve_device));
    if(!extra_data) free(fibs);
}


__global__ void kernel_set_model_inital_conditions(real *sv, int num_volumes)
{
    // Thread ID
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if(threadID < num_volumes) {

        *((real*)((char*)sv + pitch * 0) + threadID) = INITIAL_V;   // V;       millivolt
        *((real*)((char*)sv + pitch * 1) + threadID) = 0.005619; //M
        *((real*)((char*)sv + pitch * 2) + threadID) = 0.551265; //H
        *((real*)((char*)sv + pitch * 3) + threadID) = 0.246963; //J
        *((real*)((char*)sv + pitch * 4) + threadID) = 0.060715; //Xr1
        *((real*)((char*)sv + pitch * 5) + threadID) = 0.018069; //Xs
        *((real*)((char*)sv + pitch * 6) + threadID) = 0.201403; //S
        *((real*)((char*)sv + pitch * 7) + threadID) = 0.842582; //F
        *((real*)((char*)sv + pitch * 8) + threadID) = 0.997207; //F2
        *((real*)((char*)sv + pitch * 9) + threadID) = 0.000072; //D_INF
        *((real*)((char*)sv + pitch * 10) + threadID) = 0.0; //R_INF
        *((real*)((char*)sv + pitch * 11) + threadID) = 0.412887; //Xr2_INF
    }
}


// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(real dt, real *sv, real* stim_currents,
                          uint32_t *cells_to_solve, uint32_t num_cells_to_solve,
                          int num_steps, real *fibrosis,  real atpi,
                          real Ko, real Ki_multiplicator, real K1_multiplicator, real acidosis)
{
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        real rDY[NEQ];

        for (int n = 0; n < num_steps; ++n) {

            RHS_gpu(sv, rDY, stim_currents[threadID], sv_id, dt, fibrosis[threadID], atpi, Ko, Ki_multiplicator, K1_multiplicator, acidosis);

            *((real*)((char*)sv) + sv_id) = dt*rDY[0] + *((real*)((char*)sv) + sv_id);

            for(int i = 1; i < 12; i++) {
                *((real*)((char*)sv + pitch * i) + sv_id) = rDY[i];
            }

        }

    }
}


inline __device__ void RHS_gpu(real *sv_, real *rDY_, real stim_current, int threadID_, real dt, real fibrosis, real atpi, real Ko, real Ki_multiplicator, real K1_multiplicator, real acidosis) {
    //fibrosis = 0 means that the cell is fibrotic, 1 is not fibrotic. Anything between 0 and 1 means border zone
    const real svolt = *((real*)((char*)sv_ + pitch * 0) + threadID_);

    const real sm   = *((real*)((char*)sv_ + pitch * 1) + threadID_);
    const real sh   = *((real*)((char*)sv_ + pitch * 2) + threadID_);
    const real sj   = *((real*)((char*)sv_ + pitch * 3) + threadID_);
    const real sxr1 = *((real*)((char*)sv_ + pitch * 4) + threadID_);
    const real sxs  = *((real*)((char*)sv_ + pitch * 5) + threadID_);
    const real ss   = *((real*)((char*)sv_ + pitch * 6) + threadID_);
    const real sf  = *((real*)((char*)sv_ + pitch * 7) + threadID_);
    const real sf2  = *((real*)((char*)sv_ + pitch * 8) + threadID_);
    const real D_INF  = *((real*)((char*)sv_ + pitch * 9) + threadID_);
    const real Xr2_INF  = *((real*)((char*)sv_ + pitch * 10) + threadID_);
    const real R_INF  = *((real*)((char*)sv_ + pitch * 11) + threadID_);

    const real natp = 0.24;          // K dependence of ATP-sensitive K current
    const real nicholsarea = 0.00005; // Nichol's areas (cm^2)
    const real hatp = 2;             // Hill coefficient

    //Extracellular potassium concentration was elevated
    //from its default value of 5.4 mM to values between 6.0 and 8.0 mM
    //Ref: A Comparison of Two Models of Human Ventricular Tissue: Simulated Ischemia and Re-entry
    real Ko_change  = 5.4f - Ko;
    Ko = Ko + Ko_change*fibrosis;

    //Linear changing of atpi depending on the fibrosis and distance from the center of the scar (only for border zone cells)
    real atpi_change = 6.8f - atpi;
    atpi = atpi + atpi_change*fibrosis;

    //real katp = 0.306;
    //Ref: A Comparison of Two Models of Human Ventricular Tissue: Simulated Ischaemia and Re-entry
    //real katp = 0.306;
    const real katp = -0.0942857142857*atpi + 0.683142857143; //Ref: A Comparison of Two Models of Human Ventricular Tissue: Simulated Ischaemia and Re-entry


    const real patp =  1/(1 + pow((atpi/katp),hatp));
    const real gkatp    =  0.000195/nicholsarea;
    const real gkbaratp =  gkatp*patp*pow((Ko/4),natp);

    const real katp2= 1.4;
    const real hatp2 = 2.6;
    const real pcal = 1.0/(1.0 + pow((katp2/atpi),hatp2));


    const real Cao=2.0;
    const real Nao=140.0;
    const real Cai=0.00007;
    const real Nai=7.67;
    const real Ki=138.3;

//Constants
    const real R=8314.472;
    const real F=96485.3415;
    const real T=310.0;
    const real RTONF=(R*T)/F;

//Parameters for currents
//Parameters for IKr
    const real Gkr=0.101;
//Parameters for Iks
    const real pKNa=0.03;
#ifdef EPI
    const real Gks=0.257;
#endif
#ifdef ENDO
    const real Gks=0.392;
#endif
#ifdef MCELL
    const real Gks=0.098;
#endif
//Parameters for Ik1
    const real GK1=5.405;
//Parameters for Ito
#ifdef EPI
    const real Gto=0.294;
#endif
#ifdef ENDO
    const real Gto=0.073;
#endif
#ifdef MCELL
    const real Gto=0.294;
#endif
//Parameters for INa
    const real GNa=14.838*0.5; //ACIDOSIS
//Parameters for IbNa
    const real GbNa=0.00029;
//Parameters for INaK
    const real KmK=1.0;
    const real KmNa=40.0;
    const real knak=2.724;
//Parameters for ICaL
    const real GCaL=0.2786*pcal*0.5; //ACIDOSIS
//Parameters for IbCa
    const real GbCa=0.000592;
//Parameters for INaCa
    const real knaca=1000;
    const real KmNai=87.5;
    const real KmCa=1.38;
    const real ksat=0.1;
    const real n=0.35;
//Parameters for IpCa
    const real GpCa=0.1238;
    const real KpCa=0.0005;
//Parameters for IpK;
    const real GpK=0.0293;


    const real Ek=RTONF*(log((Ko/Ki)));
    const real Ena=RTONF*(log((Nao/Nai)));
    const real Eks=RTONF*(log((Ko+pKNa*Nao)/(Ki+pKNa*Nai)));
    const real Eca=0.5*RTONF*(log((Cao/Cai)));
    real IKr;
    real IKs;
    real IK1;
    real Ito;
    real INa;
    real IbNa;
    real ICaL;
    real IbCa;
    real INaCa;
    real IpCa;
    real IpK;
    real INaK;
    real IKatp;

    real Ak1;
    real Bk1;
    real rec_iK1;
    real rec_ipK;
    real rec_iNaK;
    real AM;
    real BM;
    real AH_1;
    real BH_1;
    real AH_2;
    real BH_2;
    real AJ_1;
    real BJ_1;
    real AJ_2;
    real BJ_2;
    real M_INF;
    real H_INF;
    real J_INF;
    real TAU_M;
    real TAU_H;
    real TAU_J;
    real axr1;
    real bxr1;
    real Xr1_INF;
    real Xr2_INF_new;
    real TAU_Xr1;
    real Axs;
    real Bxs;
    real Xs_INF;
    real TAU_Xs;
    real R_INF_new;
    real S_INF;
    real TAU_S;
    real Af;
    real Bf;
    real Cf;
    real Af2;
    real Bf2;
    real Cf2;
    real D_INF_new;
    real TAU_F;
    real F_INF;
    real TAU_F2;
    real F2_INF;
    real sItot;


    //Needed to compute currents
    Ak1=0.1/(1.+exp(0.06*(svolt-Ek-200)));
    Bk1=(3.*exp(0.0002*(svolt-Ek+100))+
         exp(0.1*(svolt-Ek-10)))/(1.+exp(-0.5*(svolt-Ek)));
    rec_iK1=Ak1/(Ak1+Bk1);
    rec_iNaK=(1./(1.+0.1245*exp(-0.1*svolt*F/(R*T))+0.0353*exp(-svolt*F/(R*T))));
    rec_ipK=1./(1.+exp((25-svolt)/5.98));


    //Compute currents
    INa=GNa*sm*sm*sm*sh*sj*((svolt-3.4)-Ena); //ACIDOSIS
    ICaL=GCaL*D_INF*sf*sf2*((svolt-3.4)-60); //ACIDOSIS
    Ito=Gto*R_INF*ss*(svolt-Ek);
    IKr=Gkr*sqrt(Ko/5.4)*sxr1*Xr2_INF*(svolt-Ek);
    IKs=Gks*sxs*sxs*(svolt-Eks);
    IK1=GK1*rec_iK1*(svolt-Ek);
    INaCa=knaca*(1./(KmNai*KmNai*KmNai+Nao*Nao*Nao))*(1./(KmCa+Cao))*
          (1./(1+ksat*exp((n-1)*svolt*F/(R*T))))*
          (exp(n*svolt*F/(R*T))*Nai*Nai*Nai*Cao-
           exp((n-1)*svolt*F/(R*T))*Nao*Nao*Nao*Cai*2.5);

    INaCa = INaCa*0.6; //ACIDOSIS

    INaK=knak*(Ko/(Ko+KmK))*(Nai/(Nai+KmNa))*rec_iNaK;
    IpCa=GpCa*Cai/(KpCa+Cai);
    IpK=GpK*rec_ipK*(svolt-Ek);
    IbNa=GbNa*(svolt-Ena);
    IbCa=GbCa*(svolt-Eca);

    IKatp = gkbaratp*(svolt-Ek);


    //Determine total current
    (sItot) = IKr    +
              IKs   +
              IK1   +
              Ito   +
              INa   +
              IbNa  +
              ICaL  +
              IbCa  +
              INaK  +
              INaCa +
              IpCa  +
              IpK   +
              IKatp +
              stim_current;

    //compute steady state values and time constants
    AM=1./(1.+exp((-60.-svolt)/5.));
    BM=0.1/(1.+exp((svolt+35.)/5.))+0.10/(1.+exp((svolt-50.)/200.));
    TAU_M=AM*BM;
    M_INF=1./((1.+exp((-56.86-svolt)/9.03))*(1.+exp((-56.86-svolt)/9.03)));
    if (svolt>=-40.)
    {
        AH_1=0.;
        BH_1=(0.77/(0.13*(1.+exp(-(svolt+10.66)/11.1))));
        TAU_H= 1.0/(AH_1+BH_1);
    }
    else
    {
        AH_2=(0.057*exp(-(svolt+80.)/6.8));
        BH_2=(2.7*exp(0.079*svolt)+(3.1e5)*exp(0.3485*svolt));
        TAU_H=1.0/(AH_2+BH_2);
    }
    H_INF=1./((1.+exp((svolt+71.55)/7.43))*(1.+exp((svolt+71.55)/7.43)));
    if(svolt>=-40.)
    {
        AJ_1=0.;
        BJ_1=(0.6*exp((0.057)*svolt)/(1.+exp(-0.1*(svolt+32.))));
        TAU_J= 1.0/(AJ_1+BJ_1);
    }
    else
    {
        AJ_2=(((-2.5428e4)*exp(0.2444*svolt)-(6.948e-6)*
                                             exp(-0.04391*svolt))*(svolt+37.78)/
              (1.+exp(0.311*(svolt+79.23))));
        BJ_2=(0.02424*exp(-0.01052*svolt)/(1.+exp(-0.1378*(svolt+40.14))));
        TAU_J= 1.0/(AJ_2+BJ_2);
    }
    J_INF=H_INF;

    Xr1_INF=1./(1.+exp((-26.-svolt)/7.));
    axr1=450./(1.+exp((-45.-svolt)/10.));
    bxr1=6./(1.+exp((svolt-(-30.))/11.5));
    TAU_Xr1=axr1*bxr1;
    Xr2_INF_new=1./(1.+exp((svolt-(-88.))/24.));


    Xs_INF=1./(1.+exp((-5.-svolt)/14.));
    Axs=(1400./(sqrt(1.+exp((5.-svolt)/6))));
    Bxs=(1./(1.+exp((svolt-35.)/15.)));
    TAU_Xs=Axs*Bxs+80;

#ifdef EPI
    R_INF_new=1./(1.+exp((20-svolt)/6.));
    S_INF=1./(1.+exp((svolt+20)/5.));
    TAU_S=85.*exp(-(svolt+45.)*(svolt+45.)/320.)+5./(1.+exp((svolt-20.)/5.))+3.;
#endif
#ifdef ENDO
    R_INF_new=1./(1.+exp((20-svolt)/6.));
    S_INF=1./(1.+exp((svolt+28)/5.));
    TAU_S=1000.*exp(-(svolt+67)*(svolt+67)/1000.)+8.;
#endif
#ifdef MCELL
    R_INF_new=1./(1.+exp((20-svolt)/6.));
    S_INF=1./(1.+exp((svolt+20)/5.));
    TAU_S=85.*exp(-(svolt+45.)*(svolt+45.)/320.)+5./(1.+exp((svolt-20.)/5.))+3.;
#endif


    D_INF_new=1./(1.+exp((-8-svolt)/7.5));
    F_INF=1./(1.+exp((svolt+20)/7));
    Af=1102.5*exp(-(svolt+27)*(svolt+27)/225);
    Bf=200./(1+exp((13-svolt)/10.));
    Cf=(180./(1+exp((svolt+30)/10)))+20;
    TAU_F=Af+Bf+Cf;
    F2_INF=0.67/(1.+exp((svolt+35)/7))+0.33;
    Af2=600*exp(-(svolt+27)*(svolt+27)/170);
    Bf2=7.75/(1.+exp((25-svolt)/10));
    Cf2=16/(1.+exp((svolt+30)/10));
    TAU_F2=Af2+Bf2+Cf2;

    //update voltage
    rDY_[0] = -sItot;

    //Update gates
    rDY_[1] = M_INF-(M_INF-sm)*exp(-dt/TAU_M);
    rDY_[2] = H_INF-(H_INF-sh)*exp(-dt/TAU_H);
    rDY_[3] = J_INF-(J_INF-sj)*exp(-dt/TAU_J);
    rDY_[4] = Xr1_INF-(Xr1_INF-sxr1)*exp(-dt/TAU_Xr1);
    rDY_[5] = Xs_INF-(Xs_INF-sxs)*exp(-dt/TAU_Xs);
    rDY_[6]= S_INF-(S_INF-ss)*exp(-dt/TAU_S);
    rDY_[7] =F_INF-(F_INF-sf)*exp(-dt/TAU_F);
    rDY_[8] =F2_INF-(F2_INF-sf2)*exp(-dt/TAU_F2);

    rDY_[9] = D_INF_new;
    rDY_[10] = R_INF_new;
    rDY_[11] = Xr2_INF_new;



}
